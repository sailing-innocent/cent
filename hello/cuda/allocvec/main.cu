#include <cstdio>
#include <hip/hip_runtime.h>
#include <vector>

// eastl

template <class T>
struct CudaAllocator {
    using value_type = T;
    CudaAllocator() = default;
    template<class _Other>
    constexpr CudaAllocator(const CudaAllocator<_Other>&) noexcept {}

    T *allocate(size_t size) {
        T *ptr = nullptr;
        hipMallocManaged(&ptr, size * sizeof(T));
        return ptr;
    }

    void deallocate(T *ptr, size_t size = 0) {
        hipFree(ptr);
    }
};

__global__ void kernel(int *arr, int n) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x;
         i < n; i += blockDim.x * gridDim.x) {
        arr[i] = i;
    }
}

int main() {
    int n = 10;
    std::vector<int, CudaAllocator<int>> arr(n);
    kernel<<<2, 8>>>(arr.data(), n);
    hipDeviceSynchronize();
    for (int i = 0; i < n; i++) {
        printf("arr[%d]: %d\n", i, arr[i]);
    }

    return 0;
}